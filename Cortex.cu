#include "hip/hip_runtime.h"
#include "Cortex.cuh"
#include <iostream>
#include "CUDAHelper.cuh"

__constant__ float ALPHA;
__constant__ float SHRINK;
__constant__ uint2 CORT_IMG_SIZE;
__constant__ size_t GAUSS_KERNEL_WIDTH;
__constant__ float GAUSS_SIGMA;

struct add_double2 {
    __device__ double2 operator()(const double2& a, const double2& b) const {
        double2 r;
        r.x = a.x + b.x;
        r.y = a.y + b.y;
        return r;
    }
 };

struct min_vals_double2 {
    __device__ double2 operator()(const double2& a, const double2& b) const {
        double2 r;
        r.x = a.x < b.x ? a.x : b.x ;
        r.y = a.y < b.y ? a.y : b.y;
        return r;
    }
 };

struct max_vals_double2 {
    __device__ double2 operator()(const double2& a, const double2& b) const {
        double2 r;
        r.x = a.x < b.x ? b.x : a.x ;
        r.y = a.y < b.y ? b.y : a.y;
        return r;
    }
 };

__device__ double gauss(float sigma, float x, float y, float mean = 0.0) {
	float norm = sqrtf(x*x + y*y);
	return exp(-powf((norm - mean), 2) / (2 * powf(sigma, 2))) / sqrtf(2 * M_PI * powf(sigma, 2));
}

__global__ void cort_map_left_kernel(SamplingPoint *d_leftFields, double2 *d_leftLoc, size_t size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (size <= index)
		return;

	SamplingPoint *point = &d_leftFields[index];
	d_leftLoc[index].y = sqrtf(powf(point->_x - ALPHA, 2) + powf(point->_y, 2));
	double theta = atan2(point->_y, point->_x - ALPHA);
	d_leftLoc[index].x = theta + (theta < 0 ? M_PI : -M_PI);
}

__global__ void cort_map_right_kernel(SamplingPoint *d_rightFields, double2 *d_rightLoc, size_t size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (size <= index)
		return;

	SamplingPoint *point = &d_rightFields[index];
	d_rightLoc[index].y = sqrtf(powf(point->_x + ALPHA, 2) + powf(point->_y, 2));
	d_rightLoc[index].x = atan2(point->_y, point->_x + ALPHA);
}

__global__ void cort_image_kernel(double *d_img, double *d_norm_img, double *d_img_vector,
		SamplingPoint *d_fields, double2 *d_loc, double *d_gauss, size_t size, size_t vecLen, bool rgb) {

	int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if (size <= globalIndex)
		return;

	int channel = globalIndex / (size / (rgb ? 3 : 1));
	int offset = channel * CORT_IMG_SIZE.x * CORT_IMG_SIZE.y;
	int index = globalIndex % (size / (rgb ? 3 : 1));
	int vecOffset = channel * vecLen;

	double x = d_loc[index].x;
	double y = d_loc[index].y;

	int dx = (int)(10 * ((round(x * 10) / 10 - round(x))));
	dx < 0 ? dx = 10 + dx : dx;
	int dy = (int)(10 * ((round(y * 10) / 10 - round(y))));
	dy < 0 ? dy = 10 + dy : dy;

	double *kernel = &d_gauss[(dx * 10 + dy) * GAUSS_KERNEL_WIDTH * GAUSS_KERNEL_WIDTH];

	int X = (int)round(x) - GAUSS_KERNEL_WIDTH / 2;
	int Y = (int)round(y) - GAUSS_KERNEL_WIDTH / 2;

	double value = d_img_vector[vecOffset + d_fields[index]._i];
	for (int i = 0; i != GAUSS_KERNEL_WIDTH; ++i) {
		for (int j = 0; j != GAUSS_KERNEL_WIDTH; ++j) {
			if (X + j >= 0 && Y + i >= 0 && X + j < CORT_IMG_SIZE.x && Y + i < CORT_IMG_SIZE.y) {
				atomicAdd(&d_norm_img[offset + (Y + i) * CORT_IMG_SIZE.x + X + j], kernel[i * GAUSS_KERNEL_WIDTH + j]);
				atomicAdd(&d_img[offset + (Y + i) * CORT_IMG_SIZE.x + X + j], value * kernel[i * GAUSS_KERNEL_WIDTH + j]);
			}
		}
	}
}

__global__ void cort_prepare_kernel(double2 *d_loc, double2 min, size_t size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (size <= index)
		return;

	d_loc[index].x += GAUSS_KERNEL_WIDTH - min.x;
	d_loc[index].x *= SHRINK;

	d_loc[index].y += GAUSS_KERNEL_WIDTH - min.y;
	d_loc[index].y *= SHRINK;
}

__global__ void euclidean_distance_kernel(double2 *d_loc, double2 *d_out, size_t size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (size * size <= index)
		return;

	int x = index % size;
	int y = index / size;

	double2 a = d_loc[x];
	double2 b = d_loc[y];

	d_out[index].x = sqrtf(powf((b.x - a.x), 2));
	d_out[index].y = sqrtf(powf((b.y - a.y), 2));
}

__global__ void scale_theta_flip_y_kernel(double2 *d_loc, double norm, size_t size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (size <= index)
		return;

	d_loc[index].x *= norm;
	d_loc[index].y *= -1;
}

__global__ void gauss_kernel(double *gauss100) {
	int index = (blockIdx.x + threadIdx.x * blockDim.x ) * GAUSS_KERNEL_WIDTH * GAUSS_KERNEL_WIDTH;

	float x = blockIdx.x * 0.1;
	float y = threadIdx.x * 0.1;
	float dx = GAUSS_KERNEL_WIDTH / 2 + x;
	float dy = GAUSS_KERNEL_WIDTH / 2 + y;

	for (int i = 0; i != GAUSS_KERNEL_WIDTH; ++i) {
		for (int j = 0; j != GAUSS_KERNEL_WIDTH; ++j) {
			gauss100[index + i * GAUSS_KERNEL_WIDTH + j] = gauss(GAUSS_SIGMA, dx - i, dy - j);
		}
	}
}

__global__ void normalise(uchar *d_norm, double *d_image, double *normaliser, size_t size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (size <= index)
		return;

	d_norm[index] = normaliser[index] == 0.0 ? 0 : (int)(d_image[index] / normaliser[index]);
}

template <class T>
__host__ void setPointerToNull(T **d_ptr) {
	if (*d_ptr != nullptr){
		hipFree(*d_ptr);
		cudaCheckErrors("ERROR");
		*d_ptr = nullptr;
	}
}

Cortex::~Cortex() {
	setPointerToNull(&d_leftLoc);
	setPointerToNull(&d_rightLoc);
	setPointerToNull(&d_gauss);
}

__host__ int Cortex::cortImage(double *h_imageVector, size_t vecLen, uchar *h_result,
			size_t cortImgX, size_t cortImgY, bool rgb, double *d_imageVector,
			SamplingPoint *d_fields, double2 *d_loc, size_t size) const {
	if (!isReady())
		return ERRORS::uninitialized;
	if ((h_imageVector == nullptr && d_imageVector == nullptr) || h_result == nullptr)
		return ERRORS::invalidArguments;
	if (cortImgX != _cortImgSize.x || cortImgY != _cortImgSize.y || rgb != _rgb ||
			vecLen != _channels * (_leftCortexSize + _rightCortexSize))
		return ERRORS::imageParametersDidNotMatch;
	double *d_img;
	hipMalloc((void**)&d_img, _channels * _cortImgSize.x * _cortImgSize.y * sizeof(double));
	hipMemset(d_img, 0.0, _channels * _cortImgSize.x * _cortImgSize.y * sizeof(double));
	double *d_av_img;
	hipMalloc((void**)&d_av_img, _channels * _cortImgSize.x * _cortImgSize.y * sizeof(double));
	hipMemset(d_av_img, 0.0, _channels * _cortImgSize.x * _cortImgSize.y * sizeof(double));
	double *_d_imageVector;
	if (d_imageVector != nullptr)
		_d_imageVector = d_imageVector;
	else {
		hipMalloc((void**)&_d_imageVector, _channels * (_leftCortexSize + _rightCortexSize) * sizeof(double));
		hipMemcpy(_d_imageVector, h_imageVector, _channels * (_leftCortexSize + _rightCortexSize) * sizeof(double), hipMemcpyHostToDevice);
	}

	cort_image_kernel<<<ceil(_channels * size / 512.0), 512>>>(d_img, d_av_img, _d_imageVector,
			d_fields, d_loc, d_gauss, _channels * size, _leftCortexSize + _rightCortexSize, _rgb);
	//hipDeviceSynchronize();
	cudaCheckErrors("ERROR");

	uchar *d_norm;
	hipMalloc((void**)&d_norm, _channels * _cortImgSize.x * _cortImgSize.y * sizeof(uchar));
	normalise<<<ceil(_channels * _cortImgSize.x * _cortImgSize.y / 512.0), 512>>>(d_norm, d_img, d_av_img, _channels * _cortImgSize.x * _cortImgSize.y);
	//hipDeviceSynchronize();
	cudaCheckErrors("ERROR");

	hipMemcpy(h_result, d_norm, _channels * _cortImgSize.x * _cortImgSize.y * sizeof(uchar), hipMemcpyDeviceToHost);
	cudaCheckErrors("ERROR");

	hipFree(d_norm);
	if (d_imageVector == nullptr)
		hipFree(_d_imageVector);
	hipFree(d_av_img);
	hipFree(d_img);
	return 0;
}

__host__ int Cortex::cortImageLeft(double *h_imageVector, size_t vecLen, uchar *h_result,
							size_t cortImgX, size_t cortImgY, bool rgb, double *d_imageVector) const {
	return cortImage(h_imageVector, vecLen, h_result, cortImgX, cortImgY, rgb,
					 d_imageVector, d_leftFields, d_leftLoc, _leftCortexSize);
	/*double *d_img;
	hipMalloc((void**)&d_img, _channels * _cortImgSize.x * _cortImgSize.y * sizeof(double));
	hipMemset(d_img, 0.0, _channels * _cortImgSize.x * _cortImgSize.y * sizeof(double));
	double *d_av_img;
	hipMalloc((void**)&d_av_img, _channels * _cortImgSize.x * _cortImgSize.y * sizeof(double));
	hipMemset(d_av_img, 0.0, _channels * _cortImgSize.x * _cortImgSize.y * sizeof(double));
	double *_d_imageVector;
	hipMalloc((void**)&_d_imageVector, _channels * (_leftCortexSize + _rightCortexSize) * sizeof(double));
	hipMemcpy(_d_imageVector, h_imageVector, _channels * (_leftCortexSize + _rightCortexSize) * sizeof(double), hipMemcpyHostToDevice);

	cort_image_kernel<<<ceil(_channels * _leftCortexSize / 512.0), 512>>>(d_img, d_av_img, _d_imageVector,
			d_leftFields, d_leftLoc, d_gauss, _channels * _leftCortexSize, _leftCortexSize + _rightCortexSize, _rgb);
	//hipDeviceSynchronize();
	cudaCheckErrors("ERROR");

	uchar *d_norm;
	hipMalloc((void**)&d_norm, _channels * _cortImgSize.x * _cortImgSize.y * sizeof(uchar));
	normalise<<<ceil(_channels * _cortImgSize.x * _cortImgSize.y / 512.0), 512>>>(d_norm, d_img, d_av_img, _channels * _cortImgSize.x * _cortImgSize.y);
	//hipDeviceSynchronize();
	cudaCheckErrors("ERROR");

	hipMemcpy(h_result, d_norm, _channels * _cortImgSize.x * _cortImgSize.y * sizeof(uchar), hipMemcpyDeviceToHost);
	cudaCheckErrors("ERROR");

	hipFree(d_norm);
	hipFree(_d_imageVector);
	hipFree(d_av_img);
	hipFree(d_img);
	return 0;*/
}

__host__ int Cortex::cortImageRight(double *h_imageVector, size_t vecLen, uchar *h_result,
							size_t cortImgX, size_t cortImgY, bool rgb, double *d_imageVector) const {
	return cortImage(h_imageVector, vecLen, h_result, cortImgX, cortImgY, rgb,
					 d_imageVector, d_rightFields, d_rightLoc, _rightCortexSize);
	/*double *d_img;
	hipMalloc((void**)&d_img, _channels * _cortImgSize.x * _cortImgSize.y * sizeof(double));
	hipMemset(d_img, 0.0, _channels * _cortImgSize.x * _cortImgSize.y * sizeof(double));
	double *d_av_img;
	hipMalloc((void**)&d_av_img, _channels * _cortImgSize.x * _cortImgSize.y * sizeof(double));
	hipMemset(d_av_img, 0.0, _channels * _cortImgSize.x * _cortImgSize.y * sizeof(double));
	double *_d_imageVector;
	hipMalloc((void**)&_d_imageVector, _channels * (_leftCortexSize + _rightCortexSize) * sizeof(double));
	hipMemcpy(_d_imageVector, h_imageVector, _channels * (_leftCortexSize + _rightCortexSize) * sizeof(double), hipMemcpyHostToDevice);

	cort_image_kernel<<<ceil(_channels * _rightCortexSize / 512.0), 512>>>(d_img, d_av_img, _d_imageVector,
			d_rightFields, d_rightLoc, d_gauss, _channels * _rightCortexSize, _leftCortexSize + _rightCortexSize, _rgb);
	//hipDeviceSynchronize();
	cudaCheckErrors("ERROR");

	uchar *d_norm;
	hipMalloc((void**)&d_norm, _channels * _cortImgSize.x * _cortImgSize.y * sizeof(uchar));
	normalise<<<ceil(_channels * _cortImgSize.x * _cortImgSize.y / 512.0), 512>>>(d_norm, d_img, d_av_img, _channels * _cortImgSize.x * _cortImgSize.y);
	//hipDeviceSynchronize();
	cudaCheckErrors("ERROR");

	hipMemcpy(h_result, d_norm, _channels * _cortImgSize.x * _cortImgSize.y * sizeof(uchar), hipMemcpyDeviceToHost);
	cudaCheckErrors("ERROR");

	hipFree(d_norm);
	hipFree(_d_imageVector);
	hipFree(d_av_img);
	hipFree(d_img);
	return 0;*/
}

__host__ error Cortex::locationsFromCortexFields(SamplingPoint *h_leftFields, size_t leftSize,
											SamplingPoint *h_rightFields, size_t rightSize) {

	if (isnan(_shrink) || isnan(_alpha))
		return ERRORS::uninitialized;

	setLeftCortexFields(h_leftFields, leftSize);
	setRightCortexFields(h_rightFields, rightSize);

	if (d_leftFields == nullptr || d_rightFields == nullptr)
		return ERRORS::invalidArguments;

	setPointerToNull(&d_leftLoc);
	hipMalloc((void**)&d_leftLoc, _leftCortexSize * sizeof(double2));
	cort_map_left_kernel<<<ceil(_leftCortexSize / 512.0), 512>>>(d_leftFields, d_leftLoc, _leftCortexSize);
	//hipDeviceSynchronize();
	//cudaCheckErrors("ERROR");

	setPointerToNull(&d_rightLoc);
	hipMalloc((void**)&d_rightLoc, _rightCortexSize * sizeof(double2));
	cort_map_right_kernel<<<ceil(_rightCortexSize / 512.0), 512>>>(d_rightFields, d_rightLoc, _rightCortexSize);
	hipDeviceSynchronize();
	cudaCheckErrors("ERROR");

	double2 *d_eucl_left;
	hipMalloc((void**)&d_eucl_left, _leftCortexSize * _leftCortexSize * sizeof(double2));
	euclidean_distance_kernel<<<ceil(_leftCortexSize * _leftCortexSize / 512.0), 512>>>(d_leftLoc, d_eucl_left, _leftCortexSize);
	//hipDeviceSynchronize();
	//cudaCheckErrors("ERROR");

	double2 *d_eucl_right;
	hipMalloc((void**)&d_eucl_right, _rightCortexSize * _rightCortexSize * sizeof(double2));
	euclidean_distance_kernel<<<ceil(_rightCortexSize * _rightCortexSize / 512.0), 512>>>(d_rightLoc, d_eucl_right, _rightCortexSize);
	hipDeviceSynchronize();
	cudaCheckErrors("ERROR");

	thrust::device_ptr<double2> d_leftLoc_begin(d_eucl_left);
	thrust::device_ptr<double2> d_leftLoc_end(d_eucl_left + _leftCortexSize * _leftCortexSize);
	thrust::device_ptr<double2> d_rightLoc_begin(d_eucl_right);
	thrust::device_ptr<double2> d_rightLoc_end(d_eucl_right + _rightCortexSize * _rightCortexSize);

	double2 init; init.x = init.y = 0.0;
	double2 sum_left = thrust::reduce(d_leftLoc_begin, d_leftLoc_end, init, add_double2());

	init.x = init.y = 0.0;
	double2 sum_right = thrust::reduce(d_rightLoc_begin, d_rightLoc_end, init, add_double2());

	double xd = (sum_left.x / (_leftCortexSize * _leftCortexSize) + sum_right.x / (_rightCortexSize * _rightCortexSize)) / 2;
	double yd = (sum_left.y / (_leftCortexSize * _leftCortexSize) + sum_right.y / (_rightCortexSize * _rightCortexSize)) / 2;

	scale_theta_flip_y_kernel<<<ceil(_leftCortexSize / 512.0), 512>>>(d_leftLoc, yd/xd, _leftCortexSize);
	//hipDeviceSynchronize();
	//cudaCheckErrors("ERROR");

	scale_theta_flip_y_kernel<<<ceil(_rightCortexSize / 512.0), 512>>>(d_rightLoc, yd/xd, _rightCortexSize);
	hipDeviceSynchronize();
	cudaCheckErrors("ERROR");

	thrust::device_ptr<double2> d_l_b(d_leftLoc);
	thrust::device_ptr<double2> d_l_e(d_leftLoc + _leftCortexSize);
	init.x = init.y = 10000.0;
	double2 min_l = thrust::reduce(d_l_b, d_l_e, init, min_vals_double2());

	thrust::device_ptr<double2> d_r_b(d_rightLoc);
	thrust::device_ptr<double2> d_r_e(d_rightLoc + _rightCortexSize);
	init.x = init.y = 10000.0;
	double2 min_r = thrust::reduce(d_r_b, d_r_e, init, min_vals_double2());

	cort_prepare_kernel<<<ceil(_leftCortexSize / 512.0), 512>>>(d_leftLoc, min_l, _leftCortexSize);
	//hipDeviceSynchronize();
	//cudaCheckErrors("ERROR");

	cort_prepare_kernel<<<ceil(_rightCortexSize / 512.0), 512>>>(d_rightLoc, min_r, _rightCortexSize);
	hipDeviceSynchronize();
	cudaCheckErrors("ERROR");

	init.x = init.y = -10000.0;
	_cortImgSize.x = thrust::reduce(d_l_b, d_l_e, init, max_vals_double2()).x + _gaussKernelWidth / 2;
	_cortImgSize.y = thrust::reduce(d_l_b, d_l_e, init, max_vals_double2()).y + _gaussKernelWidth / 2;

	hipMemcpyToSymbol(HIP_SYMBOL(CORT_IMG_SIZE), &_cortImgSize, sizeof(uint2));
	cudaCheckErrors("ERROR");

	hipFree(d_eucl_left);
	hipFree(d_eucl_right);

	return 0;
}

__host__ void Cortex::gauss100() {
	setPointerToNull(&d_gauss);
	hipMalloc((void**)&d_gauss, 100 * _gaussKernelWidth * _gaussKernelWidth * sizeof(double));
	gauss_kernel<<<10, 10>>>(d_gauss);
	hipDeviceSynchronize();
	cudaCheckErrors("ERROR");
}

__host__ bool Cortex::isReady() const {
	return  _leftCortexSize != 0 && _rightCortexSize != 0 &&
			d_leftLoc != nullptr && d_rightLoc != nullptr &&
			_cortImgSize.x != 0 && _cortImgSize.y != 0 &&
			_gaussKernelWidth != 0 && d_gauss != nullptr;
}

__host__ void Cortex::setAlpha(float alpha) {
	if (alpha == _alpha)
		return;
	setPointerToNull(&d_leftLoc);
	setPointerToNull(&d_rightLoc);
	_alpha = alpha;
	hipMemcpyToSymbol(HIP_SYMBOL(ALPHA), &_alpha, sizeof(float));
	cudaCheckErrors("ERROR");
}

__host__ void Cortex::setShrink(float shrink) {
	if (shrink == _shrink)
		return;
	setPointerToNull(&d_leftLoc);
	setPointerToNull(&d_rightLoc);
	_shrink = shrink;
	hipMemcpyToSymbol(HIP_SYMBOL(SHRINK), &_shrink, sizeof(float));
	cudaCheckErrors("ERROR");
}

__host__ void Cortex::setRGB(bool rgb) {
	if (rgb == _rgb)
		return;
	_rgb = rgb;
	_channels = _rgb ? 3 : 1;
}

__host__ void Cortex::setCortImageSize(uint2 cortImgSize) {
	if (cortImgSize.x == _cortImgSize.x && cortImgSize.y == _cortImgSize.y)
		return;
	_cortImgSize = cortImgSize;
	hipMemcpyToSymbol(HIP_SYMBOL(CORT_IMG_SIZE), &_cortImgSize, sizeof(uint2));
	cudaCheckErrors("ERROR");
}

__host__ error Cortex::getLeftCortexFields(SamplingPoint *h_leftFields, size_t leftSize) const {
	return getFromDevice(h_leftFields, leftSize, d_leftFields, _leftCortexSize);
}

__host__ error Cortex::setLeftCortexFields(const SamplingPoint *h_leftFields, const size_t leftSize) {
	return setOnDevice(h_leftFields, leftSize, &d_leftFields, _leftCortexSize);
}

__host__ error Cortex::getRightCortexFields(SamplingPoint *h_rightFields, size_t rightSize) const {
	return getFromDevice(h_rightFields, rightSize, d_rightFields, _rightCortexSize);
}

__host__ error Cortex::setRightCortexFields(const SamplingPoint *h_rightFields, size_t rightSize) {
	return setOnDevice(h_rightFields, rightSize, &d_rightFields, _rightCortexSize);
}

__host__ error Cortex::getLeftCortexLocations(double2 *h_leftLoc, size_t leftSize) const {
	return getFromDevice(h_leftLoc, leftSize, d_leftLoc, _leftCortexSize);
}

__host__ int Cortex::setLeftCortexLocations(const double2 *h_leftLoc, size_t leftSize) {
	if (leftSize != _leftCortexSize)
		return ERRORS::cortexSizeDidNotMatch;
	return setOnDevice(h_leftLoc, leftSize, &d_leftLoc, _leftCortexSize);
	/*if (h_leftLoc != nullptr) {
		if (leftSize != _leftCortexSize)
			return ERRORS::cortexSizeDidNotMatch;
		setPointerToNull(&d_leftLoc);
		hipMalloc((void**)&d_leftLoc, sizeof(double2) * _leftCortexSize);
		hipMemcpy(d_leftLoc, h_leftLoc, sizeof(double2) * _leftCortexSize, hipMemcpyHostToDevice);
		cudaCheckErrors("ERROR");
	}

	return 0;*/
}

__host__ error Cortex::getRightCortexLocations(double2 *h_rightLoc, size_t rightSize) const {
	return getFromDevice(h_rightLoc, rightSize, d_rightLoc, _rightCortexSize);
}

__host__ int Cortex::setRightCortexLocations(const double2 *h_rightLoc, size_t rightSize) {
	if (rightSize != _rightCortexSize)
		return ERRORS::cortexSizeDidNotMatch;
	return setOnDevice(h_rightLoc, rightSize, &d_rightLoc, _rightCortexSize);
	/*if (h_rightLoc != nullptr) {
		if (rightSize != _rightCortexSize)
			return ERRORS::cortexSizeDidNotMatch;
		setPointerToNull(&d_rightLoc);
		hipMalloc((void**)&d_rightLoc, sizeof(double2) * _rightCortexSize);
		hipMemcpy(d_rightLoc, h_rightLoc, sizeof(double2) * _rightCortexSize, hipMemcpyHostToDevice);
		cudaCheckErrors("ERROR");
	}

	return 0;*/
}

__host__ error Cortex::getGauss100( double *h_gauss, size_t kernelWidth, float sigma) const {
	if (kernelWidth != _gaussKernelWidth || sigma != _gaussSigma)
		return ERRORS::invalidArguments;
	hipMemcpy(h_gauss, d_gauss, 100 * _gaussKernelWidth * _gaussKernelWidth * sizeof(double), hipMemcpyDeviceToHost);
	cudaCheckErrors("ERROR");
	return 0;
}

__host__ error Cortex::setGauss100(const size_t kernelWidth, const float sigma, double *h_gauss) {
	if (kernelWidth == 0)
		return ERRORS::invalidArguments;
	_gaussKernelWidth = kernelWidth;
	hipMemcpyToSymbol(HIP_SYMBOL(GAUSS_KERNEL_WIDTH), &_gaussKernelWidth, sizeof(size_t));
	_gaussSigma = sigma;
	hipMemcpyToSymbol(HIP_SYMBOL(GAUSS_SIGMA), &_gaussSigma, sizeof(float));
	cudaCheckErrors("ERROR");

	if (h_gauss == nullptr) {
		gauss100();
	} else {
		setPointerToNull(&d_gauss);
		hipMalloc((void**)&d_gauss, 100 * _gaussKernelWidth * _gaussKernelWidth * sizeof(double));
		hipMemcpy(d_gauss, h_gauss, 100 * _gaussKernelWidth * _gaussKernelWidth * sizeof(double), hipMemcpyHostToDevice);
		cudaCheckErrors("ERROR");
	}
	return 0;
}

template <class T>
	__host__ error Cortex::getFromDevice(T *h_ptr, const size_t h_size, const T *d_ptr, const size_t d_size) const {
	if (h_ptr == nullptr || h_size == 0)
		return ERRORS::invalidArguments;
	if (h_size != d_size)
		return ERRORS::cortexSizeDidNotMatch;
	if (d_ptr == nullptr)
		return ERRORS::uninitialized;
	hipMemcpy(h_ptr, d_ptr, sizeof(T) * d_size, hipMemcpyDeviceToHost);
	cudaCheckErrors("ERROR");
	return 0;
}

template <class T>
	__host__ error Cortex::setOnDevice(const T *h_ptr, size_t h_size, T **d_ptr, size_t &d_size) {
	if (h_ptr == nullptr || h_size == 0)
		return ERRORS::invalidArguments;

	setPointerToNull(d_ptr);
	hipMalloc((void**)d_ptr, sizeof(T) * h_size);
	hipMemcpy(*d_ptr, h_ptr, sizeof(T) * h_size, hipMemcpyHostToDevice);
	d_size = h_size;
	cudaCheckErrors("ERROR");
	return 0;
}
